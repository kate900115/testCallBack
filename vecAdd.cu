#include "hip/hip_runtime.h"
#include "vecAdd.h"

extern "C" __global__ void vecAdd(int *a, int *b, int* c, void* p){
	int* flag = (int*)p;
	
	int count=0;
/*	while(count!=100){
		count++;
		clock_t start = clock();
		clock_t now;
		for (;;){
			now = clock();
			clock_t cycles = now > start? now - start: now+(0xffffffff - start);
			if (cycles >= 100000000){
				break;
			}
		}
	}
*/
	int tid = blockIdx.x;
	if (tid < N){
		c[tid]=a[tid]+b[tid];
	}

	*flag = 0;
	printf("after computation flag = %d\n", *flag);
	

}
